#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void vec_add(float *A, float *B, float *C, int N){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (i >= N) {return;}
	
	C[i] = A[i] + B[i];
}

void main(){

	int N = 200

	float *A_h = new float[N];
	float *B_h = new float[N];
	float *C_h = new float[N];

	for(int i0; i<N; i++){
		A_h[i] = 1.3f;
		B_h[i] = 2.0f;
	}

	float *A_d, *B_d, *C_d;

	hipMalloc( (void**) &A_d, N * sizeof(float));
	hipMalloc( (void**) &B_d, N * sizeof(float));
	hipMalloc( (void**) &C_d, N * sizeof(float));

	hipMemcpy(A_d, A_h, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, N*sizeof(float), hipMemcpyHostToDevice);

	int blocks = int(N-0.5)/256 + 1;
	vec_add<<<blocks, 256>>> (A_d, B_d, C_d, N);

	hipMemcpy(C_h, C_d, N*sizeof(float), hipMemcpyDeviceToHost)

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

}
